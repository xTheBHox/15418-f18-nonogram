//
// Created by Benjamin Huang on 11/19/2018.
//

#include "Board2DDevice.h"

Board2DDevice *board2d_init_host(unsigned w, unsigned h) {

    // Allocate the board header
    Board2DDevice *B = (Board2DDevice *)malloc(sizeof(Board2DDevice));

    if (B == NULL) {
        fprintf(stderr, "Failed to allocate board header\n");
        return NULL;
    }

    B->w = w;
    B->h = h;
    B->dirty = true;
    B->valid = true;
    B->solved = false;

    // Allocate the board data array

    size_t b_len = w * h;
    B->data = (NonogramColor *)calloc(2 * b_len, sizeof(NonogramColor));

    if (B->data == NULL) {
        fprintf(stderr, "Failed to allocate board data array\n");
        free(B);
        return NULL;
    }

    B->dataCM = &B->data[b_len];
    return B;

}

#ifdef __NVCC__
__inline__ unsigned nextPow2(unsigned n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

Board2DDevice *board2d_init_dev(Board2DDevice *B_host) {

    Board2DDevice B_tmp_var;
    Board2DDevice *B_tmp = &B_tmp_var;
    void *B_dev;

    board2d_dev_init_copy(B_tmp, B_host);
    B_tmp->pitchRM = nextPow2(B_host->w);
    B_tmp->pitchCM = nextPow2(B_host->h);
#ifdef DEBUG
    std::cout << "pitchRM: " << B_tmp->pitchRM << std::endl;
    std::cout << "pitchCM: " << B_tmp->pitchCM << std::endl;
#endif

    cudaCheckError(hipMalloc((void **)&(B_tmp->data), sizeof(NonogramColor) * B_tmp->pitchRM * B_host->h));
    cudaCheckError(hipMemcpy2D(
        (void *)B_tmp->data, sizeof(NonogramColor) * B_tmp->pitchRM,
        (void *)B_host->data, sizeof(NonogramColor) * B_host->w,
        sizeof(NonogramColor) * B_host->w, B_host->h, hipMemcpyHostToDevice
    ));

    cudaCheckError(hipMalloc((void **)&B_tmp->dataCM, sizeof(NonogramColor) * B_tmp->pitchCM * B_host->w));
    cudaCheckError(hipMemcpy2D(
        (void *)B_tmp->dataCM, sizeof(NonogramColor) * B_tmp->pitchCM,
        (void *)B_host->dataCM, sizeof(NonogramColor) * B_host->h,
        sizeof(NonogramColor) * B_host->h, B_host->w, hipMemcpyHostToDevice
    ));

    cudaCheckError(hipMalloc(&B_dev, sizeof(Board2DDevice)));
    cudaCheckError(hipMemcpy(B_dev, (void *)B_tmp, sizeof(Board2DDevice), hipMemcpyHostToDevice));

    B_host->pitchRM = B_tmp->pitchRM;
    B_host->pitchCM = B_tmp->pitchCM;

    return (Board2DDevice *)B_dev;

}
#endif

void board2d_free_host(Board2DDevice *B) {

    free(B->data);
    free(B);

}

#ifdef __NVCC__
void board2d_cleanup_dev(Board2DDevice *B_host, Board2DDevice *B_dev) {
    Board2DDevice B_tmp_var;
    Board2DDevice *B_tmp = &B_tmp_var;

    cudaCheckError(hipMemcpy(
        (void *)B_tmp, (void *)B_dev, sizeof(Board2DDevice), hipMemcpyDeviceToHost
    ));

    cudaCheckError(hipMemcpy2D(
        (void *)B_host->data, B_host->w,
        (void *)B_tmp->data, B_host->pitchRM,
        B_host->w, B_host->h, hipMemcpyDeviceToHost
    ));
    cudaCheckError(hipMemcpy2D(
        (void *)B_host->dataCM, B_host->h,
        (void *)B_tmp->dataCM, B_host->pitchCM,
        B_host->h, B_host->w, hipMemcpyDeviceToHost
    ));

    cudaCheckError(hipFree((void *)B_tmp->data));
    cudaCheckError(hipFree((void *)B_dev));

}
#endif

Board2DDevice *board2d_deepcopy_host(Board2DDevice *B) {

    Board2DDevice *B_copy = board2d_init_host(B->w, B->h);
    memcpy((void *)B_copy->data, (void *)B->data, 2 * B->w * B->h * sizeof(NonogramColor));
    B_copy->solved = B->solved;
    B_copy->valid = B->valid;
    B_copy->dirty = true;
    return B_copy;

}

std::ostream &operator<<(std::ostream &os, Board2DDevice *B) {

    for (unsigned r = 0; r < B->h; r++) {
        for (unsigned c = 0; c < B->w; c++) {
            char sym = 'X';
            switch (board2d_host_elem_get_rm(B, c, r)) {
                case NGCOLOR_BLACK: {
                    sym = '#';
                    break;
                }
                case NGCOLOR_UNKNOWN: {
                    sym = '?';
                    break;
                }
                case NGCOLOR_WHITE: {
                    sym = '.';
                    break;
                }
                default: {
                    sym = 'H';
                    break;
                }
            }
            os << sym;
        }
        os << std::endl;
    }
    return os;

}

__host__ __device__
void board2d_dev_init_copy(Board2DDevice *B_dst, const Board2DDevice *B_src) {
    B_dst->w = B_src->w;
    B_dst->h = B_src->h;
    B_dst->pitchRM = B_src->pitchRM;
    B_dst->pitchCM = B_src->pitchCM;
    B_dst->dirty = B_src->dirty;
    B_dst->solved = B_src->solved;
    B_dst->valid = B_src->valid;
}

__device__
void board2d_dev_mutableonly_copy(Board2DDevice *B_dst, const Board2DDevice *B_src) {
    B_dst->dirty = B_src->dirty;
    B_dst->solved = B_src->solved;
    B_dst->valid = B_src->valid;
}
